#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>

#define SIZE 128

//kernel funciton for prefix wihtmultiple threads
__gloabl__
void prefixsum(int *in, int *out){
        int xindex= threadIdx.x

        //add up all elements for that speicfic index
        for (int i = 0; i<xindex; i++){
                int val = 0
                for (int j = 0, j<=i; j++){
                        val=inp[j];
                }
                out[xindex]=val;
        }
        return 0;
}

int main(void){
        int * input, * output;
        hipMallocManaged(&input, sizeof(int)*SIZE);
        hipMallocManaged(&output, sizeof(int)*SIZE);

        //init input array
        for (int i=0; i<SIZE; i++){
                input[i]= 1;
        }
        // call prefixsum
        prefixsum<<<1,SIZE>>>(input,output);

        //sync
        hipDeviceSynchronize();
        printf("%s\n", hipGetErrorString(hipGetLastError()));

        //check errors
        float maxError = 0;
        for(int i=0; i<SIZE; i++){
                maxError=fmax(maxError,fabs(output[i]-output[i]));
        }
        std::cout << "Max error: " << maxError << std::endl ;

        hipFree(input);
        hipFree(output);
}
