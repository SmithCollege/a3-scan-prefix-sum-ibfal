#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <sys/time.h>
#define SIZE 128

double get_clock() {
	struct timeval tv; int ok;
	ok = gettimeofday(&tv, (void *) 0);
	if (ok<0) { printf(“gettimeofday error”); }
	return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
	}


//kernel funciton for prefix wihtmultiple threads
__gloabl__
void prefixsum(int *in, int *out){
        int xindex= threadIdx.x

        //add up all elements for that speicfic index
        for (int i = 0; i<xindex; i++){
                int val = 0
                for (int j = 0, j<=i; j++){
                        val=inp[j];
                }
                out[xindex]=val;
        }
        return 0;
}

int main(void){
		double t0 = get_clock();
		for (i=0; i<N; i++) times[i] = get_clock();
		double t1 = get_clock();
		printf("time per call: %f ns\n", (1000000000.0*(t1-t0)/N) );
				
        int * input, * output;
        hipMallocManaged(&input, sizeof(int)*SIZE);
        hipMallocManaged(&output, sizeof(int)*SIZE);

        //init input array
        for (int i=0; i<SIZE; i++){
                input[i]= 1;
        }
        // call prefixsum
       	double start = get_clock();
        prefixsum<<<1,SIZE>>>(input,output);

        //sync
        hipDeviceSynchronize();
        
        double end = get_clock()
		printf("start: %f  end: %f", start, end);
        
        printf("%s\n", hipGetErrorString(hipGetLastError()));

        //check errors
        float maxError = 0;
        for(int i=0; i<SIZE; i++){
                maxError=fmax(maxError,fabs(output[i]-output[i]));
        }
        std::cout << "Max error: " << maxError << std::endl ;

        hipFree(input);
        hipFree(output);
}
