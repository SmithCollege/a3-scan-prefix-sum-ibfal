#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <sys/time.h>
#define SIZE 8

double get_clock() {
	struct timeval tv; int ok;
	ok = gettimeofday(&tv, (void *) 0);
	if (ok<0) { printf(“gettimeofday error”); }
	return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
	}

//kernel funciton for prefix wihtmultiple threads
__global__ 
void prefixsum(int *in, int *out, int stride){
		int xindex = threadIdx.x;
        //int index = blockIdx.x*blockDim.x+threadIdx.x;
        
        if (xindex >= stride){
  			out[xindex] = in[xindex]+ in[xindex - stride];
  			}		
  			
   		if (xindex < stride){
        	out[xindex]= in[xindex];}
        }

int main(void){
		double t0 = get_clock();
		for (i=0; i<N; i++) times[i] = get_clock();
		double t1 = get_clock();
		printf("time per call: %f ns\n", (1000000000.0*(t1-t0)/N) );
		
        int *input, *output, *temp, *source, *dest;
        hipMallocManaged(&input, sizeof(int)*SIZE);
        hipMallocManaged(&output, sizeof(int)*SIZE);
        //hipMallocManaged(&temp, sizeof(int)*SIZE);
        //hipMallocManaged(&dest, sizeof(int)*SIZE);
        //hipMallocManaged(&source, sizeof(int)*SIZE);

        //init input array
		input[0]= 3;
		input[1]= 1;
		input[2]= 7;
		input[3]= 0;
		input[4]= 4;
		input[5]= 1;
		input[6]= 6;
		input[7]= 3;

		source = &input[0];
		dest = &output[0];
		
        for (int i = 0; i<SIZE; i++){
        		printf("%d ", input[i]);}
        printf("\n");

        
        double start = get_clock();
        // call prefixsum
        for (int stride = 1; stride < SIZE; stride*=2){
        	prefixsum<<<1,SIZE>>>(source, dest, stride);
        	temp = dest; 
        	output = source;
        	source = temp ;
		}
	
        //sync
        hipDeviceSynchronize();
        double end = get_clock();
        printf("start: %f  end: %f", start, end);
        
        printf("%s\n", hipGetErrorString(hipGetLastError()));

		for (int i = 0; i<SIZE; i++){
			printf("%d ", output[i]);}

        //check errors
        float maxError = 0;
       	for(int i=0; i<SIZE; i++){
        	maxError=fmax(maxError,fabs(output[i]-output[i]));
                }
        std::cout << "Max error: " << maxError << std::endl ;
        
        hipFree(input);
        hipFree(output);
        
        }
