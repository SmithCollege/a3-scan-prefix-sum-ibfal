
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

#define SIZE 8

//kernel funciton for prefix wihtmultiple threads
__global__ 
void prefixsum(int *in, int *out, int stride){
		int xindex = threadIdx.x;
        //int index = blockIdx.x*blockDim.x+threadIdx.x;
        
        if (xindex >= stride){
  			out[xindex] = in[xindex]+ in[xindex - stride];
  			}		
  			
   		if (xindex < stride){
        	out[xindex]= in[xindex];}
        }

int main(void){
        int *input, *output, *temp, *source, *dest;
        hipMallocManaged(&input, sizeof(int)*SIZE);
        hipMallocManaged(&output, sizeof(int)*SIZE);
        //cudaMallocManaged(&temp, sizeof(int)*SIZE);
        //cudaMallocManaged(&dest, sizeof(int)*SIZE);
        //cudaMallocManaged(&source, sizeof(int)*SIZE);

        //init input array
		input[0]= 3;
		input[1]= 1;
		input[2]= 7;
		input[3]= 0;
		input[4]= 4;
		input[5]= 1;
		input[6]= 6;
		input[7]= 3;

		source = &input[0];
		dest = &output[0];
		
        for (int i = 0; i<SIZE; i++){
        		printf("%d ", input[i]);}
        printf("\n");
        
        // call prefixsum
        for (int stride = 1; stride < SIZE; stride*=2){
        	prefixsum<<<1,SIZE>>>(source, dest, stride);
        	temp = dest; 
        	output = source;
        	source = temp ;
		}
	
        //sync
        hipDeviceSynchronize();
        printf("%s\n", hipGetErrorString(hipGetLastError()));

		for (int i = 0; i<SIZE; i++){
			printf("%d ", output[i]);}

        //check errors
        float maxError = 0;
       	for(int i=0; i<SIZE; i++){
        	maxError=fmax(maxError,fabs(output[i]-output[i]));
                }
        std::cout << "Max error: " << maxError << std::endl ;
        
        hipFree(input);
        hipFree(output);
        
        }
