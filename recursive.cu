
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <sys/time.h>
#define SIZE 1024

double get_clock() {
	struct timeval tv; int ok;
	ok = gettimeofday(&tv, (void *) 0);
	if (ok<0) {printf("gettimeofday error"); }
	return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
	}

//kernel funciton for prefix wihtmultiple threads
__global__ 
void prefixsum(int *in, int *out, int stride){
		int xindex = threadIdx.x;
        //int index = blockIdx.x*blockDim.x+threadIdx.x;
        
        if (xindex >= stride){
  			out[xindex] = in[xindex]+ in[xindex - stride];
  			}		
  			
   		if (xindex < stride){
        	out[xindex]= in[xindex];}
        }

int main(void){
		int *times;
		hipMallocManaged(&times, sizeof(double)*SIZE);
		double t0 = get_clock();
		for (int i=0; i<SIZE; i++) 
			times[i] = get_clock();
		double t1 = get_clock();
		printf("time per call: %f ns\n", (1000000000.0*(t1-t0)/SIZE) );
		
        int *input, *output, *temp, *source, *dest;
        hipMallocManaged(&input, sizeof(int)*SIZE);
        hipMallocManaged(&output, sizeof(int)*SIZE);

		for (int i = 0; i<SIZE; i++){
			input[i]=1;}

		source = &input[0];
		dest = &output[0];

		
        for (int i = 0; i<SIZE; i++){
        		printf("%d ", input[i]);}
        printf("\n");

        
        double start = get_clock();
        // call prefixsum
        for (int stride = 1; stride < SIZE; stride*=2){
        	prefixsum<<<1,SIZE>>>(source, dest, stride);
        	temp = dest; 
        	output = source;
        	source = temp ;
		}
	
        //sync
        hipDeviceSynchronize();
        double end = get_clock();
        printf("start: %f  end: %f", start, end);
        
        printf("%s\n", hipGetErrorString(hipGetLastError()));

		for (int i = 0; i<SIZE; i++){
			printf("%d ", output[i]);}

        //check errors
        float maxError = 0;
       	for(int i=0; i<SIZE; i++){
        	maxError=fmax(maxError,fabs(output[i]-output[i]));
                }
        std::cout << "Max error: " << maxError << std::endl ;
        
        hipFree(input);
        hipFree(output);
        
        }
