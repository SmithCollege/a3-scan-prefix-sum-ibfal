
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

#define SIZE 128

//kernel funciton for prefix wihtmultiple threads
__global__ 
void prefixsum(int *in, int *out){
		__shared__ int dest[SIZE];
		int xindex = threadIdx.x;
        int index = blockIdx.x*blockDim.x+threadIdx.x;
        
		if (index< SIZE){
			dest[xindex]= in[index];
		}

        for (int stride = 1; stride< blockDim.x; stride++){
        	__syncthreads();
			int val; 
        	if (xindex >= stride){
  				val= in[xindex - stride];}
			__syncthreads();
			if (xindex >=stride){
				dest[xindex]+= val;}		
        }
        out[index]= dest[xindex];
        }

int main(void){
        int * input, * output;
        hipMallocManaged(&input, sizeof(int)*SIZE);
        hipMallocManaged(&output, sizeof(int)*SIZE);

        //init input array
        for (int i=0; i<SIZE; i++){
                input[i]= 1;
        }

        for (int i = 0; i<SIZE; i++){
        		printf("%d ", input[i]);}
        printf("\n");
        
        // call prefixsum
        prefixsum<<<1,SIZE>>>(input,output);

	
        //sync
        hipDeviceSynchronize();
        printf("%s\n", hipGetErrorString(hipGetLastError()));

		for (int i = 0; i<SIZE; i++){
			printf("%d ", output[i]);}

        //check errors
        float maxError = 0;
       	for(int i=0; i<SIZE; i++){
        	maxError=fmax(maxError,fabs(output[i]-output[i]));
                }
        std::cout << "Max error: " << maxError << std::endl ;
        
        hipFree(input);
        hipFree(output);
        
        }
